#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <signal.h>
#include <exception>

#define X_RES 2048
#define Y_RES 2048

__device__ float ray_sphere_intersect(float cx, float cy, float cz, float clipx, float clipy, float clipz, float spherex, float spherey, float spherez, float spherer)
{
    float a = clipx*clipx+clipy*clipy+clipz*clipz;
    float b = 2.0f*(clipx*(cx-spherex)+clipy*(cy-spherey)+clipz*(cz-spherez));
    float c = cx*cx+spherex*spherex+cy*cy+spherey*spherey+cz*cz+spherez*spherez-2.0f*cx*spherex-2.0f*cy*spherey-2.0f*cz*spherez-spherer*spherer;
    float discriminant = b*b-4.0f*a*c;
    if (discriminant < 0) {
        return 0.0f;
    }
    else {
        return (-b - sqrtf(discriminant)) / (2.0f*a);
    }
}

__device__ float unit_x(float x, float y, float z)
{
    float mag = sqrtf(x*x+y*y+z*z);
    return x/mag;
}

__device__ float unit_y(float x, float y, float z)
{
    float mag = sqrtf(x*x+y*y+z*z);
    return y/mag;
}

__device__ float unit_z(float x, float y, float z)
{
    float mag = sqrtf(x*x+y*y+z*z);
    return z/mag;
}

__device__ float clamp(float x, float a, float b)
{
    return fmaxf(a, fminf(b, x));
}

__global__ void stress(float *a, float *objects, int numobjects)
{
    while (true)
    {
        int idx = ((blockIdx.y*blockDim.y+threadIdx.y)*blockDim.x*gridDim.x+(blockIdx.x*blockDim.x+threadIdx.x))*3;

        float numxsamples = 20.0f;
        float numysamples = 20.0f;
        float numsamples = numxsamples*numysamples;

        float cx = 0.0f;
        float cy = 0.0f;
        float cz = 1.0f;

        float lightx = 10.0f;
        float lighty = 6.0f;
        float lightz = 0.0f;
        float lightint = 1.0f; // light intensity

        for (int xs = 0; xs<=numxsamples; xs++) {
            for (int ys = 0; ys<=numysamples; ys++) {
                float clipx = (blockIdx.x*blockDim.x+threadIdx.x)/16.0f - 8.0f + (xs-numxsamples/2.0f)*0.1f/numxsamples;
                float clipy = (blockIdx.y*blockDim.y+threadIdx.y)/16.0f - 8.0f + (ys-numysamples/2.0f)*0.1f/numysamples;
                float clipz = -50.0f; // clipping plane distance

                // Ray Coloring
                float r = 1.0f-blockIdx.y/512.0f;
                float g = 1.0f-blockIdx.y/512.0f;
                float b = 1.0f;

                float sphere_dist;
                float spherex, spherey, spherez, spherer, sphere_col_r, sphere_col_g, sphere_col_b;

                bool processed = false;

                for (int i = 0; i < numobjects; i++) {
                
                    spherex = objects[i*7];
                    spherey = objects[i*7+1];
                    spherez = objects[i*7+2];
                    spherer = objects[i*7+3];
                    sphere_col_r = objects[i*7+4];
                    sphere_col_g = objects[i*7+5];
                    sphere_col_b = objects[i*7+6];

                    float sd = ray_sphere_intersect(cx, cy, cz, clipx, clipy, clipz, spherex, spherey, spherez, spherer);

                    if (0.0f < sd) {
                        if (!processed) { // pixel still empty?
                            sphere_dist = sd;
                            processed = true;
                        }
                        if (sd <= sphere_dist) {
                            sphere_dist = sd;

                            // Calculate surface normal <n>
                            float nx = cx+sphere_dist*clipx-spherex;
                            float ny = cy+sphere_dist*clipy-spherey;
                            float nz = cz+sphere_dist*clipz-spherez;
                            nx = unit_x(nx, ny, nz);
                            ny = unit_y(nx, ny, nz);
                            nz = unit_z(nx, ny, nz);

                            float tlx = lightx-nx; // from point to light x
                            float tly = lighty-ny;
                            float tlz = lightz-nz;
                            tlx = unit_x(tlx, tly, tlz);
                            tly = unit_y(tlx, tly, tlz);
                            tlz = unit_z(tlx, tly, tlz);

                            float light_dot = ((nx*tlx+ny*tly+nz*tlz-1.0f)/-2.0f)*lightint;

                            r = sphere_col_r*light_dot;
                            g = sphere_col_g*light_dot;
                            b = sphere_col_b*light_dot;
                        }
                    }
                }
            a[idx] += r;
            a[idx+1] += g;
            a[idx+2] += b;
            }
        }
        a[idx] = clamp(a[idx]/numsamples, 0.0f, 1.0f)*255.0f;
        a[idx+1] = clamp(a[idx+1]/numsamples, 0.0f, 1.0f)*255.0f;
        a[idx+2] = clamp(a[idx+2]/numsamples, 0.0f, 1.0f)*255.0f;
    }
}

int main() {

    printf("Setting up...\n");

    int cudaDeviceCount = 0;

    hipGetDeviceCount(&cudaDeviceCount);

    printf("Located %i GPUS to use...\n", cudaDeviceCount);

    float *d_a, *d_b;
    const int len = X_RES*Y_RES*3;
    float *a = new float[len];

    float b[] = {-1.0f, 0.0f, -1.3f, 0.8f, 0.8f,  0.8f,  0.75f,
                0.0f,  -0.8f, -1.3f, 1.0f, 0.64f, 0.11f, 0.08f,
                1.0f,   0.0f, -1.3f, 0.8f, 0.8f,  0.8f,  0.75f};

    int numobjects = 3;

    dim3 numBlocks(256, 256, 1);
    dim3 threadsPerBlock(32, 32, 1);

    hipMalloc((void**)&d_a, len*sizeof(float));
    hipMalloc((void**)&d_b, numobjects*7*sizeof(float));

    hipMemcpy(d_a, a, len*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, numobjects*7*sizeof(float), hipMemcpyHostToDevice);

    printf("Starting D2O GPU Test! (Press ctrl+c to terminate)\n");

    for (int i = 0; i<cudaDeviceCount; i++) { // Stress test for multiple GPUS currently unsupported
        hipSetDevice(i);
        printf("Starting GPU Test on GPU %i\n", i);
        stress<<<numBlocks, threadsPerBlock>>>(d_a, d_b, numobjects);
    }
    hipDeviceSynchronize();

    // we're never gonna get here :/
    hipMemcpy(a, d_a, len*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);

    free(a);
    free(b);

    printf("Finished D2O GPU Test!\n");

    return 0;
}